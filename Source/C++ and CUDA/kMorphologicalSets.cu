#include "hip/hip_runtime.h"
/**
kMorphologicalSets.cu
Purpose: Cluster data using the k-Morphological Sets algorithm on the GPU. 
It contains the actual algorithm as a whole, with GPU and CPU functions in it.

@author �rick Oliveira Rodrigues
@version 1.0 9/9/2016
*/

#include <stdio.h>
#include <math.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

//to call from .cpp
#include "Common.h"

//a constant for the background value in the images
#define C -1

//setting the index of your CUDA device
#define CUDA_DEVICE_INDEX 0 

#define SQUARED 1
#define DIAMOND 2
#define CROSS 3
#define CROSS_S 4
#define SQUARED_10 5

//define the chosen structuring element
#define STRUCT_ELEMENT SQUARED //it can be SQUARED, SQUARED_10, DIAMOND, CROSS or CROSS_S. Different structuring elements can be implemented as well.

//this structuring element refers to how the image is dilated before the clusterization is performed
#define STRUCT_ELEMENT_PREV SQUARED //it can be SQUARED, DIAMOND, CROSS or CROSS_S. Different structuring elements can be implemented as well.

//indexes each and every pixel of a input boolean image
__global__ void
indexImg(const bool* __restrict__ mask, int* __restrict__ data, const int WIDTH, const int HEIGHT){
	const int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < WIDTH*HEIGHT){
		if (mask[id]) data[id] = id;
		else data[id] = C;
	}
}

//kernel for dilating an input boolean/binary image
__global__ void
dilate(const bool* __restrict__ inputImg, const int WIDTH, const int HEIGHT, bool* __restrict__ outputImg){
	const int id = blockDim.x * blockIdx.x + threadIdx.x;
	const int leap = 1;
	
	const int cX = id % WIDTH, cY = id / WIDTH;
	if (cX >= 0 && cX < WIDTH && cY >= 0 && cY < HEIGHT){
#if STRUCT_ELEMENT_PREV == SQUARED
		if (cX - leap >= 0) if (inputImg[id - leap] > 0) outputImg[id] = true;
		if (cX + leap < WIDTH) if (inputImg[id + leap] > 0) outputImg[id] = true;
		if (cY - leap >= 0) if (inputImg[id - WIDTH*leap] > 0) outputImg[id] = true;
		if (cY + leap < HEIGHT) if (inputImg[id + WIDTH*leap] > 0) outputImg[id] = true;
		//digonals
		if (cX - leap >= 0 && cY - leap >= 0) if (inputImg[id - WIDTH*leap - leap] > 0) outputImg[id] = true;
		if (cY - leap >= 0 && cX + leap < WIDTH) if (inputImg[id - WIDTH*leap + leap] > 0) outputImg[id] = true;
		if (cY + leap < HEIGHT && cX - leap >= 0) if (inputImg[id + WIDTH*leap - leap] > 0) outputImg[id] = true;
		if (cY + leap < HEIGHT && cX + leap < WIDTH) if (inputImg[id + WIDTH*leap + leap] > 0) outputImg[id] = true;
#elif STRUCT_ELEMENT_PREV == DIAMOND
		if (cX - leap >= 0) if (inputImg[id - leap] > 0) outputImg[id] = true;
		if (cX + leap < WIDTH) if (inputImg[id + leap] > 0) outputImg[id] = true;
		if (cY - leap >= 0) if (inputImg[id - WIDTH*leap] > 0) outputImg[id] = true;
		if (cY + leap < HEIGHT) if (inputImg[id + WIDTH*leap] > 0) outputImg[id] = true;
#elif STRUCT_ELEMENT_PREV == CROSS
		if (cX - leap >= 0 && cY - leap >= 0) if (inputImg[id - WIDTH*leap - leap] > 0) outputImg[id] = true;
		if (cY - leap >= 0 && cX + leap < WIDTH) if (inputImg[id - WIDTH*leap + leap] > 0) outputImg[id] = true;
		if (cY + leap < HEIGHT && cX - leap >= 0) if (inputImg[id + WIDTH*leap - leap] > 0) outputImg[id] = true;
		if (cY + leap < HEIGHT && cX + leap < WIDTH) if (inputImg[id + WIDTH*leap + leap] > 0) outputImg[id] = true;
#elif STRUCT_ELEMENT_PREV == CROSS_S
		if (cX - leap >= 0) if (inputImg[id - leap] > 0) outputImg[id] = true;
		if (cX + leap < WIDTH) if (inputImg[id + leap] > 0) outputImg[id] = true;
		if (cY - leap >= 0) if (inputImg[id - WIDTH*leap] > 0) outputImg[id] = true;
		if (cY + leap < HEIGHT) if (inputImg[id + WIDTH*leap] > 0) outputImg[id] = true;
		//digonals
		if (cX - leap >= 0 && cY - leap >= 0) if (inputImg[id - WIDTH*leap - leap] > 0) outputImg[id] = true;
		if (cY - leap >= 0 && cX + leap < WIDTH) if (inputImg[id - WIDTH*leap + leap] > 0) outputImg[id] = true;
		if (cY + leap < HEIGHT && cX - leap >= 0) if (inputImg[id + WIDTH*leap - leap] > 0) outputImg[id] = true;
		if (cY + leap < HEIGHT && cX + leap < WIDTH) if (inputImg[id + WIDTH*leap + leap] > 0) outputImg[id] = true;
		//
		if (cX - leap*3 >= 0 && cY - leap*3 >= 0) if (inputImg[id - WIDTH*leap*3 - leap*3] > 0) outputImg[id] = true;
		if (cY - leap*3 >= 0 && cX + leap*3 < WIDTH) if (inputImg[id - WIDTH*leap*3 + leap*3] > 0) outputImg[id] = true;
		if (cY + leap*3 < HEIGHT && cX - leap*3 >= 0) if (inputImg[id + WIDTH*leap*3 - leap*3] > 0) outputImg[id] = true;
		if (cY + leap*3 < HEIGHT && cX + leap*3 < WIDTH) if (inputImg[id + WIDTH*leap*3 + leap*3] > 0) outputImg[id] = true;
#endif
	}
}


//Dilates and verifies if the processing ended.
//This function is somewhat random, it may return slightly different solutions from run to run.
//However, we have tested it and in our experiments the results were always the same.
//The randomness can be mitigated if more memory is consumed to duplicate the data array.
__global__ void
dilateAndVerify(int* __restrict__ data/*img*/, volatile int* __restrict__ kArray, int *finished, 
int *idempotence, const int leap, const int WIDTH, const int HEIGHT, const int K)
{
	const int id = blockDim.x * blockIdx.x + threadIdx.x;
	int cValue = 0, old;
	int previousValue = 0;
	bool still = true;

	const int cX = id % WIDTH, cY = id / WIDTH;
	cValue = data[id];
	previousValue = cValue;
	if (id < WIDTH*HEIGHT && cValue != C){
		
		//dilate according to the chosen structuring element
#if STRUCT_ELEMENT == SQUARED
		//dilates the image
		if (cX - leap >= 0) if (data[id - leap] > cValue) cValue = data[id - leap];
		if (cX + leap < WIDTH) if (data[id + leap] > cValue) cValue = data[id + leap];
		if (cY - leap >= 0) if (data[id - WIDTH*leap] > cValue) cValue = data[id - WIDTH*leap];
		if (cY + leap < HEIGHT) if (data[id + WIDTH*leap] > cValue) cValue = data[id + WIDTH*leap];
		//digonals
		if (cX - leap >= 0 && cY - leap >= 0) if (data[id - WIDTH*leap - leap] > cValue) cValue = data[id - WIDTH*leap - leap];
		if (cY - leap >=0 && cX + leap < WIDTH) if (data[id - WIDTH*leap + leap] > cValue) cValue = data[id - WIDTH*leap + leap];
		if (cY + leap < HEIGHT && cX - leap >= 0) if (data[id + WIDTH*leap - leap] > cValue) cValue = data[id + WIDTH*leap - leap];
		if (cY + leap < HEIGHT && cX + leap < WIDTH) if (data[id + WIDTH*leap + leap] > cValue) cValue = data[id + WIDTH*leap + leap];
#elif STRUCT_ELEMENT == DIAMOND
		//dilates the image
		if (cX - leap >= 0) if (data[id - leap] > cValue) cValue = data[id - leap];
		if (cX + leap < WIDTH) if (data[id + leap] > cValue) cValue = data[id + leap];
		if (cY - leap >= 0) if (data[id - WIDTH*leap] > cValue) cValue = data[id - WIDTH*leap];
		if (cY + leap < HEIGHT) if (data[id + WIDTH*leap] > cValue) cValue = data[id + WIDTH*leap];
#elif STRUCT_ELEMENT == CROSS
		if (cX - leap >= 0 && cY - leap >= 0) if (data[id - WIDTH*leap - leap] > cValue) cValue = data[id - WIDTH*leap - leap];
		if (cY - leap >= 0 && cX + leap < WIDTH) if (data[id - WIDTH*leap + leap] > cValue) cValue = data[id - WIDTH*leap + leap];
		if (cY + leap < HEIGHT && cX - leap >= 0) if (data[id + WIDTH*leap - leap] > cValue) cValue = data[id + WIDTH*leap - leap];
		if (cY + leap < HEIGHT && cX + leap < WIDTH) if (data[id + WIDTH*leap + leap] > cValue) cValue = data[id + WIDTH*leap + leap];
#elif STRUCT_ELEMENT == CROSS_S
		if (cX - leap >= 0) if (data[id - leap] > cValue) cValue = data[id - leap];
		if (cX + leap < WIDTH) if (data[id + leap] > cValue) cValue = data[id + leap];
		if (cY - leap >= 0) if (data[id - WIDTH*leap] > cValue) cValue = data[id - WIDTH*leap];
		if (cY + leap < HEIGHT) if (data[id + WIDTH*leap] > cValue) cValue = data[id + WIDTH*leap];
		//diag
		if (cX - leap >= 0 && cY - leap >= 0) if (data[id - WIDTH*leap - leap] > cValue) cValue = data[id - WIDTH*leap - leap];
		if (cY - leap >= 0 && cX + leap < WIDTH) if (data[id - WIDTH*leap + leap] > cValue) cValue = data[id - WIDTH*leap + leap];
		if (cY + leap < HEIGHT && cX - leap >= 0) if (data[id + WIDTH*leap - leap] > cValue) cValue = data[id + WIDTH*leap - leap];
		if (cY + leap < HEIGHT && cX + leap < WIDTH) if (data[id + WIDTH*leap + leap] > cValue) cValue = data[id + WIDTH*leap + leap];
		//more diags
		leap *= 3;
		if (cX - leap >= 0 && cY - leap >= 0) if (data[id - WIDTH*leap - leap] > cValue) cValue = data[id - WIDTH*leap - leap];
		if (cY - leap >= 0 && cX + leap < WIDTH) if (data[id - WIDTH*leap + leap] > cValue) cValue = data[id - WIDTH*leap + leap];
		if (cY + leap < HEIGHT && cX - leap >= 0) if (data[id + WIDTH*leap - leap] > cValue) cValue = data[id + WIDTH*leap - leap];
		if (cY + leap < HEIGHT && cX + leap < WIDTH) if (data[id + WIDTH*leap + leap] > cValue) cValue = data[id + WIDTH*leap + leap];
#elif STRUCT_ELEMENT == SQUARED_10
		//dilates the image
		if (cX - leap >= 0) if (data[id - leap] > cValue) cValue = data[id - leap];
		if (cX + leap < WIDTH) if (data[id + leap] > cValue) cValue = data[id + leap];
		if (cY - leap >= 0) if (data[id - WIDTH*leap] > cValue) cValue = data[id - WIDTH*leap];
		if (cY + leap < HEIGHT) if (data[id + WIDTH*leap] > cValue) cValue = data[id + WIDTH*leap];
		//digonals
		if (cX - leap*10 >= 0 && cY - leap*10 >= 0) if (data[id - WIDTH*leap*10 - leap*10] > cValue) cValue = data[id - WIDTH*leap*10 - leap*10];
		if (cY - leap*10 >= 0 && cX + leap*10 < WIDTH) if (data[id - WIDTH*leap*10 + leap*10] > cValue) cValue = data[id - WIDTH*leap*10 + leap*10];
		if (cY + leap*10 < HEIGHT && cX - leap*10 >= 0) if (data[id + WIDTH*leap*10 - leap*10] > cValue) cValue = data[id + WIDTH*leap*10 - leap*10];
		if (cY + leap*10 < HEIGHT && cX + leap*10 < WIDTH) if (data[id + WIDTH*leap*10 + leap*10] > cValue) cValue = data[id + WIDTH*leap*10 + leap*10];
#endif
		
		//cValue = dilatePixel(data, leap, cValue, id, cX, cY, WIDTH, HEIGHT);
		
		//synchronizes the threads in the group at this point so that it theoretically diminishes the randomness
		//__syncthreads(); //diminuir o erro com uma Imgm s�

		//updates the value for the pixel in data
		data[id] = cValue;

		
		//if (idempotence){//speeding up, early break
			if (cValue != previousValue){//idempotencia, um diferente, n�o atingiu
				atomicAnd(idempotence, false);
			}


			//if (finished){//speeding up, early break
				//rapidly and innacurately checks if ended
				still = true;
				for (int k = 0; k < K; k++){
					if (kArray[k] == cValue) {
						still = false;
						break;
					}
				}
				//more properly checks if ended
				for (int k = 0; k < K && still; k++){
					//if (kArray[k] == C) {//ou true no booleano
					old = atomicCAS((int*)&kArray[k], C, cValue); //duas threads ao mesmo tempo?

					if (old == C || kArray[k] == cValue){
						still = false;
						k = K; //out of loop
						__threadfence(); //necessary
					}
					//}
				}
				//if there is still something to do then finished is false
				if (still) atomicAnd(finished, false);
				//__threadfence();

				//checks idempotence
			//}
		//}
	}


}


/**
Auxiliar private function for printing cuda errors.
*/
inline void
print(hipError_t err, char* msg){
	if (err != hipSuccess)
	{
		printf("Error on %s ", msg);
		fprintf(stderr, "Error code: %s!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

//private global kArray
int *kArray;



/**
* Host main routine
*/
int*
kMorphologicalSets(const bool* __restrict img, const int WIDTH, const int HEIGHT, const int K, const bool disposeKArray)
{
	//error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	//setting the sizes
	const size_t DATA_SIZE = WIDTH*HEIGHT*sizeof(int),
		K_ARRAY_SIZE = K*sizeof(int),
		FINISHED_SIZE = sizeof(int),
		IMG_SIZE = WIDTH*HEIGHT*sizeof(bool);

	//kernel variables, in respect to the device at CUDA_DEVICE_INDEX constant
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, CUDA_DEVICE_INDEX); //device index = 0, you can change it if you have more CUDA devices
	const int threadsPerBlock = props.maxThreadsPerBlock / 2;
	const int blocksPerGrid = (HEIGHT*WIDTH + threadsPerBlock - 1) / threadsPerBlock;


	//allocate boolean matrix (mask)
	bool *d_mask = NULL;
	err = hipMalloc((void **)&d_mask, IMG_SIZE);
	print(err, "Failed to allocate device vector d_mask!\n");

	//copy the data to this boolean matrix
	err = hipMemcpy(d_mask, img, IMG_SIZE, hipMemcpyHostToDevice);
	print(err, "Failed to copy vector d_img from host to device!\n");


	//allocate the grey image (data)
	int *d_data = NULL;
	err = hipMalloc((void **)&d_data, DATA_SIZE);
	print(err, "Failed to allocate device vector d_data!\n");


	//call first Kernel, populate data, indexes every pixel uniquely
	indexImg << <blocksPerGrid, threadsPerBlock >> >
		(d_mask, d_data, WIDTH, HEIGHT);

	
	//erase the boolean img
	hipFree(d_mask);

	//allocate the vector kArray
	int *d_kArray = NULL;
	err = hipMalloc((void **)&d_kArray, K_ARRAY_SIZE);
	print(err, "Failed to allocate device vector d_kArray!");


	//allocate the kArrayAux
	int *d_kArrayAux = NULL;
	err = hipMalloc((void **)&d_kArrayAux, K_ARRAY_SIZE);
	print(err, "Failed to allocate device vector d_kArrayAux!");
	

	//allocate the variable finished
	int *d_finished = NULL;
	err = hipMalloc((void **)&d_finished, FINISHED_SIZE);
	print(err, "Failed to allocate device vector d_finished!");


	//allocate the device variable idempotence
	int *d_idempotence = NULL;
	err = hipMalloc((void **)&d_idempotence, FINISHED_SIZE);
	print(err, "Failed to allocate device vector d_idempotence!");

	
	//Host variables
	kArray = (int*)malloc(K_ARRAY_SIZE);
	for (int k = 0; k < K; k++) kArray[k] = C; 
	int finished = false;
	int *data = (int*)malloc(DATA_SIZE);



	//copy k array to device
	err = hipMemcpy(d_kArray, kArray, K_ARRAY_SIZE, hipMemcpyHostToDevice);
	print(err, "Failed to copy vector d_kArray from host to device!");
	

	int iterations = 0;
	int idempotence = false;
	bool idempotenceAuxVar = false;
	int leap = 1;
	int lastLeap = 1;
	// Launch the Vector Add CUDA Kernel
	//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	while (!(idempotence && finished)){//while not idempotent and not finished (i.e., while there are more than k clusters and it is not idempotent)
		//reset the cuda variables at each kernel launch
		hipMemset(d_finished, true, FINISHED_SIZE);
		hipMemset(d_idempotence, true, FINISHED_SIZE);
		hipMemset(d_kArray, C, K_ARRAY_SIZE);

		//dilate and verify if finished
		dilateAndVerify << <blocksPerGrid, threadsPerBlock>> >
			(d_data, d_kArray, d_finished, d_idempotence, leap, WIDTH, HEIGHT, K);
		print(hipGetLastError(), "Failed to launch kernel diloteAndVerify!");
			

		//Copy finished back to host
		err = hipMemcpy(&finished, d_finished, FINISHED_SIZE, hipMemcpyDeviceToHost);
		print(err, "Failed to copy vector d_finished from device to host!");
	

		//Copy idempotence back to host
		err = hipMemcpy(&idempotence, d_idempotence, FINISHED_SIZE, hipMemcpyDeviceToHost);
		print(err, "Failed to copy vector d_idempotence from device to host!");


		if (idempotence) {
			idempotenceAuxVar = true;
			if (lastLeap <= leap) leap++;
			else leap = lastLeap;
		}
		if (idempotenceAuxVar && !idempotence){
			lastLeap = leap;
			leap = 1;
		}

		iterations++;
	}

	

	//copying back the clustered image, this part can be removed for faster processing
	int* outImg = (int*)malloc(DATA_SIZE);
	err = hipMemcpy(outImg, d_data, DATA_SIZE, hipMemcpyDeviceToHost);
	print(err, "Failed to copy vector outImg from device to host!");


	//copying the kArray back
	err = hipMemcpy(kArray, d_kArray, K_ARRAY_SIZE, hipMemcpyDeviceToHost);
	print(err, "Failed to copy vector kArray from device to host!");

	//counting found clusters
	int totalClusters = 0;
	for (int k = 0; k < K; k++){
		if (kArray[k] != C) totalClusters++;
	}
	printf("The clustering ended with %d iterations and %d valid clusters. \n", iterations, totalClusters);
	printf("Cluster values/identifiers: [");
	for (int k = 0; k < K - 1; k++){
		printf("%d, ", kArray[k]);
	}
	printf("%d]\n", kArray[K - 1]);
	
	print(err, "Error on copying kArray back to host memory!");


	//free device global memory
	err = hipFree(d_data);
	print(err, "Failed to free device vector d_data!");
	err = hipFree(d_kArray);
	print(err, "Failed to free device vector d_kArray!");
	err = hipFree(d_finished);
	print(err, "Failed to free device d_idempotence!");
	err = hipFree(d_idempotence);

	//free host memory
	if (disposeKArray) free(kArray);

	//reset device
	err = hipDeviceReset();
	print(err, "Failed to deinitialize the device!");


	return outImg;
}

int*
kMorphologicalSets(const bool* __restrict img, const int WIDTH, const int HEIGHT, const int K, int* kArray)
{
	return kMorphologicalSets(img, WIDTH, HEIGHT, K, false);
}

int*
kMorphologicalSets(const bool* __restrict img, const int WIDTH, const int HEIGHT, const int K)
{
	return kMorphologicalSets(img, WIDTH, HEIGHT, K, true);
}


/**
Dilates an input boolean image and returns a boolean dilated version.

@return the pointer to the dilated image.
*/
bool*
dilateImg(const bool* __restrict inputImg, const int timesToDilate, const int WIDTH, const int HEIGHT){
	size_t SIZE = WIDTH*HEIGHT*sizeof(bool);
	hipError_t err;

	bool *outputImg = (bool*) malloc(SIZE);
	bool *d_inputImg, *d_outputImg;

	err = hipMalloc(&d_outputImg, SIZE);
	print(err, "Error on allocating d_outputImg in device memory.");

	err = hipMalloc(&d_inputImg, SIZE);
	print(err, "Error on allocating d_inputImg in device memory.");

	err = hipMemcpy(d_inputImg, inputImg, SIZE, hipMemcpyHostToDevice);
	print(err, "Error on copying inputImg to device memory.");

	err = hipMemcpy(d_outputImg, d_inputImg, SIZE, hipMemcpyDeviceToDevice);
	print(err, "Error on copying d_outputImg from d_intputImg in device memory.");


	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, CUDA_DEVICE_INDEX); //device index = 0, you can change it if you have more CUDA devices
	const int threadsPerBlock = props.maxThreadsPerBlock / 2;
	const int blocksPerGrid = (HEIGHT*WIDTH + threadsPerBlock - 1) / threadsPerBlock;

	//dilate
	int counter = 0;
	while (counter < timesToDilate){
		dilate << <blocksPerGrid, threadsPerBlock >> >
			(d_inputImg, WIDTH, HEIGHT, d_outputImg);

		err = hipMemcpy(d_inputImg, d_outputImg, SIZE, hipMemcpyDeviceToDevice);
		print(err, "Error on copying d_outputImg to d_inputImg.");

		counter++;
	}

	err = hipMemcpy(outputImg, d_outputImg, SIZE, hipMemcpyDeviceToHost);
	print(err, "Error on copying d_outputImg to host.");


	hipFree(d_inputImg);
	hipFree(d_outputImg);

	return outputImg;
}




